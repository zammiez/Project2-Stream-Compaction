#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

__device__ int O_o ()
{
	int sdfs = threadIdx.x;
	int tt;
	tt = 0;
	tt += sdfs;
	tt = 2;
	//just for debug
	//http://stackoverflow.com/questions/21911059/could-not-resolve-name-when-debug-cuda-kernel
	return tt;
}

namespace StreamCompaction {
	namespace Efficient {

		// TODO: __global__

		/**
		 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
		 */
		int * dev_o;

		__global__ void kernUpSweep(int pow2d, int * dev_odata)
		{
			int k = threadIdx.x;
			dev_odata[k * 2 * pow2d + (int)pow2d * 2 - 1] += dev_odata[k * 2 * pow2d + (int)pow2d - 1];
		}
		
		__global__ void kernDownSweep(int pow2d, int * dev_odata,int n)
		{
			int k = threadIdx.x * 2 * pow2d;

			//dev_odata[k * 2 * pow2d + (int)pow2d * 2 - 1] += dev_odata[k * 2 * pow2d + (int)pow2d - 1];
			int t = dev_odata[k + pow2d - 1];
			dev_odata[k + pow2d - 1] = dev_odata[k + pow2d * 2 - 1];
			dev_odata[k + pow2d * 2 - 1] += t;
		
		}
		__global__ void setRootZero(int * dev_odata,int n)
		{
			dev_odata[n - 1] = 0;
		}

		void initArrays(int n, const int *hst_idata)
		{
			int size = n*sizeof(int);

			hipMalloc((void**)&dev_o, size);
			//checkCUDAError("hipMalloc dev_o failed");
			hipMemcpy(dev_o, hst_idata, size, hipMemcpyHostToDevice);
			//checkCUDAError("hipMemcpy odata->dev_o failed");

		}

		void freeArrays()
		{
			hipFree(dev_o);
		}

		void scan(int n, int *odata, const int *idata) {
			// TO_DOne
			int N = ilog2ceil(n);
			N = pow(2,N);
			initArrays(N, idata);
			for (int d = 0; d <= ilog2ceil(N) - 1; d++)
			{
				int pow2d = pow(2, d);
				int end = (N - 1) / (2 * pow2d)+1;
				kernUpSweep<<<1,end>>>(pow2d,dev_o);//later:blocksize,gridsize
				/*for (int k = 0; k <= (n - 1) /( 2 * pow2d); k ++ )
				{
					x[k*2*pow2d + (int)pow2d * 2] += x[k2*pow2d + (int)pow2d - 1];
				}*/
				/*
				hipMemcpy(odata, dev_o, N*sizeof(int),hipMemcpyDeviceToHost);
				printf("\n****** d=%d\t(up)\n----[", d);
				for (int i = 0; i < N; i++)
				{
					printf(" %3d", odata[i]);
				}
				printf("]\n");*/
			}
			setRootZero<<<1,1>>>(dev_o, N);
			for (int d = ilog2ceil(N) - 1; d >= 0; d--)
			{
				int pow2d = pow(2, d);
				int end = (N - 1) / (2 * pow2d) + 1;
				kernDownSweep <<<1, end >>>(pow2d, dev_o,N);
				/*
				hipMemcpy(odata, dev_o, N*sizeof(int), hipMemcpyDeviceToHost);
				printf("\n****** d=%d\t(down)\n----[", d);
				for (int i = 0; i < N; i++)
				{
					printf(" %3d", odata[i]);
				}
				printf("]\n");*/
			}
			hipMemcpy(odata, dev_o, N*sizeof(int), hipMemcpyDeviceToHost);
			freeArrays();
			//printf("TODO\n");
		}

		/**
		 * Performs stream compaction on idata, storing the result into odata.
		 * All zeroes are discarded.
		 *
		 * @param n      The number of elements in idata.
		 * @param odata  The array into which to store elements.
		 * @param idata  The array of elements to compact.
		 * @returns      The number of elements remaining after compaction.
		 */
		int * dev_temp;
		int * dev_scan;
		int * dev_compactOut;
		int * dev_input;

		void freeCompArrays()
		{
			hipFree(dev_temp);
			hipFree(dev_scan);
			hipFree(dev_compactOut);
			hipFree(dev_input);
		}

		__global__ void kernCalcTemp(int * dev_idata,int *dev_outTemp,int n)
		{
			int index = threadIdx.x;
			if (dev_idata[index] != 0 && index<n) 
				dev_outTemp[index] = 1;
			else 
				dev_outTemp[index] = 0;
			
		}
		__global__ void kernScatter(int *dev_t,int *dev_s,int *dev_in,int *dev_outCompact)
		{
			int index = threadIdx.x;
			if (dev_t[index] == 1)
			{
				dev_outCompact[dev_s[index]] = dev_in[index];
			}
		}

		int compact(int n, int *odata, const int *idata) {
			
			int N = ilog2ceil(n);
			N = pow(2, N);
			//int N = n;

			int size = N*sizeof(int);

			hipMalloc((void**)&dev_temp, size);

			hipMalloc((void**)&dev_input, size);
			hipMemcpy(dev_input, idata, size, hipMemcpyHostToDevice);

			hipMalloc((void**)&dev_scan, size);
			

			//Step 1 : Compute temporary array.
			kernCalcTemp <<<1, N >>>(dev_input,dev_temp,n);//later:blocksize,gridsize 
			hipMemcpy(dev_scan, dev_temp, size, hipMemcpyDeviceToDevice);

			//Step 2 : Run exclusive scan

			for (int d = 0; d <= ilog2ceil(N) - 1; d++)
			{
				int pow2d = pow(2, d);
				int end = (N - 1) / (2 * pow2d) + 1;
				kernUpSweep <<<1, end >>>(pow2d, dev_scan);//later:blocksize,gridsize
			}

			setRootZero <<<1, 1 >>>(dev_scan, N);

			for (int d = ilog2ceil(N) - 1; d >= 0; d--)
			{
				int pow2d = pow(2, d);
				int end = (N - 1) / (2 * pow2d) + 1;
				kernDownSweep <<<1, end >>>(pow2d, dev_scan, N);
			}

			//Step 3 : Scatter
			int compactLength;
			hipMemcpy(&compactLength, &(dev_scan[N - 1]), sizeof(int), hipMemcpyDeviceToHost);
			hipMalloc((void**)&dev_compactOut, compactLength*sizeof(int));

			kernScatter<<<1,N>>>(dev_temp,dev_scan,dev_input,dev_compactOut);//later:blocksize,gridsize

			//hipMemcpy(odata,dev_compactOut,compactLength*sizeof(int),hipMemcpyDeviceToHost);
			
			hipMemcpy(odata, dev_compactOut, compactLength*sizeof(int), hipMemcpyDeviceToHost);
			// TO_DOne
			freeCompArrays();
			return compactLength;
		}

	}
}
