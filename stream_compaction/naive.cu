#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
	namespace Naive {

		// TODO: __global__

		/**
		 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
		 */
		int * dev_o;

		__global__ void kernNaiveScan(int pow2d_1, int *dev_odata)
		{
			int k = threadIdx.x;
			if (k >= pow2d_1)
				dev_odata[k] = dev_odata[k - pow2d_1] + dev_odata[k];
		}

		void initArrays(int n, const int *hst_idata)
		{
			int size = n*sizeof(int);

			hipMalloc((void**)&dev_o, size);
			//checkCUDAError("hipMalloc dev_o failed");
			hipMemcpy(dev_o, hst_idata, size, hipMemcpyHostToDevice);
			//checkCUDAError("hipMemcpy odata->dev_o failed");

		}

		void freeArrays()
		{
			hipFree(dev_o);
		}

		void scan(int n, int *odata, const int *idata) {
			//??? inclusive or exclusive ? not exactly 39.2/slides
			initArrays(n, idata);
			for (int d = 1; d <= ilog2ceil(n); d++)
			{
				int pow2_d_1 = pow(2, d - 1);
				kernNaiveScan <<<1, n-1 >>>(pow2_d_1,dev_o);
				/*hipMemcpy(odata, dev_o, n*sizeof(int),hipMemcpyDeviceToHost);
				printf("\nd=%d\n---[", d);
				for (int i = 0; i < n; i++)
				{
					printf("\t%d", odata[i]);
				}
				printf("]\n");*/
			}
			// TODO
			hipMemcpy(odata, dev_o, n*sizeof(int), hipMemcpyDeviceToHost);
			//inclusive to exclusive
			for (int i = n-1; i >0; i--)
			{
				odata[i] = odata[i - 1];
			}
			odata[0] = 0;
			freeArrays();
		}

	}
}
